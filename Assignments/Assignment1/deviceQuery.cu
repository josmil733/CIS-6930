#include <iostream>
#include "cudaCheck.cuh"



int main (int argc, char *argv[])
{

  int numDevices;
  cudaCheck( hipGetDeviceCount(&numDevices));
  std::cout << "Number of devices: " << numDevices << std::endl;

 

  for (int devId = 0; devId < numDevices; devId++){
     std::cout << "-------------------------------------\n";
     hipDeviceProp_t prop;
     cudaCheck( hipGetDeviceProperties(&prop, devId));
     std::cout << "Device id: " << devId << std::endl;
     std::cout << "Device name: " << prop.name << std::endl;

/*@  Calculate and print the theoretical peak bandwidth of the
     CUDA device.  You will need to obtain additional information
     from prop.  See the documentation 
*/

  }
  std::cout << "-------------------------------------\n";

}

 
