#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaCheck.cuh"
#include <stdio.h>

int main (int argc, char *argv[])
{

  int numDevices;
  cudaCheck( hipGetDeviceCount(&numDevices));
  // std::cout << "Number of devices: " << numDevices << std::endl;

 

  for (int devId = 0; devId < numDevices; devId++){
     std::cout << "-------------------------------------\n";
     hipDeviceProp_t prop;
     cudaCheck( hipGetDeviceProperties(&prop, devId));
     std::cout << "Device id: " << devId << std::endl;
     std::cout << "Device name: " << prop.name << std::endl;
     std::cout << "-------------------------------------\n";

     std::cout << "Values needed to calculate theoretical peak memory bandwidth (TPMB) (printed from hipDeviceProp_t):" << std::endl;
     std::cout << "Memory clock rate (GHz): " << prop.memoryClockRate/1000000 << std::endl;
     std::cout << "Global memory bus width (bits/transfer): " << prop.memoryBusWidth << std::endl;
     std::cout << "(TPMB) Calculation (assuming GDDR6 memory type):\n" << std::endl;
     std::cout << "TPMB = memory clock rate x (global memory bus width/8) x transfers per clock cycle\n" << std::endl;
     printf("    = %i x 10^9 cycles/second x %i B/transfer x 2 transfers/cycle\n", prop.memoryClockRate/1000000, prop.memoryBusWidth/8);
     printf("    = %i GB/s\n", (prop.memoryClockRate/1000000)*(prop.memoryBusWidth/8)*2);

/*     std::cout << "    = 7e9 cycles/second x (352/8) B/transfer x 2 transfers/cycle\n" << std::endl;
     std::cout << "    = 616 GB/s" << std::endl;
*/

//printf("%f.2", prop.memoryClockRate);

   //cudaCheck( hipDeviceGetAttribute( &max_threads_block,
          //hipDeviceAttributeMaxThreadsPerBlock ,devId) );


  }
  std::cout << "-------------------------------------\n";

}

 
