#include <iostream>
#include "cudaCheck.cuh"

int main (int argc, char *argv[])
{

  int numDevices;
  cudaCheck( hipGetDeviceCount(&numDevices));
  // std::cout << "Number of devices: " << numDevices << std::endl;

 

  for (int devId = 0; devId < numDevices; devId++){
     std::cout << "-------------------------------------\n";
     hipDeviceProp_t prop;
     cudaCheck( hipGetDeviceProperties(&prop, devId));
     std::cout << "Device id: " << devId << std::endl;
     std::cout << "Device name: " << prop.name << std::endl;
   int max_threads_block = 1;
   cudaCheck( hipDeviceGetAttribute( &max_threads_block,
          hipDeviceAttributeMaxThreadsPerBlock ,devId) );
  // cudaCheck( hipDeviceGetAttribute( &max_threads_block,

    std::cout << "Other device properties:\n";
    std::cout << "Max threads/block: " << max_threads_block << std::endl;
    std::cout << "2_Max threads/block: " << prop.maxThreadsPerBlock << std::endl;
//    std::cout << "Max threads/dim:" << prop.maxThreadsDim << std::endl;

/*@  Calculate and print the theoretical peak bandwidth of the
     CUDA device.  You will need to obtain additional information
     from prop.  See the documentation 
*/

  }
  std::cout << "-------------------------------------\n";

}

 
